
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 2048;

__global__ void add_complex(int *a , int *b , int *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    while (tid < N)
    {          
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main (void)
{
    int a[N], b[N], c[N];
    for (int i = 0; i < N; ++i)
    {
        a[i] = -i;
        b[i] = i*i; 
    }

    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void **)&dev_a, N*sizeof(int));
    hipMalloc((void **)&dev_b, N*sizeof(int));
    hipMalloc((void **)&dev_c, N*sizeof(int));

    hipMemcpy (dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy (dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("Results for <<<%d, %d>>>:\n", (N+127)/128, 128);

    hipEventRecord(start, 0);

    add_complex<<<(N+127)/128, 128>>>(dev_a, dev_b, dev_c);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float worktime;
    hipEventElapsedTime(&worktime, start, stop);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    printf("%d + %d = %d\n", a[0], b[0], c[0]);
    printf("%d + %d = %d\n", a[N - 1], b[N - 1], c[N - 1]);

    printf("Time = %f ms \n", worktime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    system("pause");
    return 0 ;
}