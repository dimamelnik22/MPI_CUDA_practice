
#include <hip/hip_runtime.h>
#include "stdio.h"

#define N   128

__global__ void add(int *A, int *B, int *C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < N)
    {
        C[i * N + j] = A[i * N + j] + B[i * N + j];
    }
}

int main( void )
{
    int a[N * N], b[N * N], c[N * N];
    int *dev_a, *dev_b, *dev_c;

    for (int i = 0; i < N * N; ++i) 
    {
        a[i] = -i;
        b[i] = i * i;
    }

    hipMalloc((void**)&dev_a, N * N * sizeof(int));
    hipMalloc((void**)&dev_b, N * N * sizeof(int));
    hipMalloc((void**)&dev_c, N * N * sizeof(int));

    hipMemcpy(dev_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N / 16, N / 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    add<<<numBlocks, threadsPerBlock>>>(dev_a, dev_b, dev_c);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float worktime;

    hipEventElapsedTime(&worktime, start, stop);

    printf("Time = %f ms \n", worktime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dev_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    printf("%d + %d = %d\n", a[0], b[0], c[0]);
    printf("%d + %d = %d\n", a[N * N - 1], b[N * N - 1], c[N * N - 1]);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
